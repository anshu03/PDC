#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void gpu_matrixadd(int *a,int *b, int *c, int N) {

	int col = threadIdx.x + blockDim.x * blockIdx.x; 
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	int index = row * N + col;

      	if(col < N && row < N)
          c[index] = a[index]+b[index];

}

void cpu_matrixadd(int *a,int *b, int *c, int N) {

	int index;
	for(int col=0;col < N; col++) 
		for(int row=0;row < N; row++) {
			index = row * N + col;
           		c[index] = a[index]+b[index];
		}
}

int main(int argc, char *argv[])  {

	char key;

	int i, j;

	int Grid_Dim_x=1, Grid_Dim_y=1;
	int Block_Dim_x=1, Block_Dim_y=1;

	int noThreads_x, noThreads_y;
	int noThreads_block;

	int N = 10;
	int *a,*b,*c,*d;
	int *dev_a, *dev_b, *dev_c;
	int size;

	hipEvent_t start, stop; 
	float elapsed_time_ms;


do {

	__global__ void input_parameter(sizeof(x), sizeof(y), nub_block))

		x = (int*) malloc(size);
		y = (int*) malloc(size);
		nub_block = (int*) malloc(size);

	for(i=0;i < N;i++)
	for(j=0;j < N;j++) {
		a[i * N + j] = i;
		b[i * N + j] = i;
	}

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a , size ,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b , size ,hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c , size ,hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
//	hipEventSynchronize(start);  	// Needed?

	gpu_matrixadd<<<Grid,Block>>>(dev_a,dev_b,dev_c,N);

	hipMemcpy(c,dev_c, size ,hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

//	for(i=0;i < N;i++) 
//	for(j=0;j < N;j++)
//	   printf("%d+%d=%d\n",a[i * N + j],b[i * N + j],c[i * N + j]);
	printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms);  // print out execution time


	hipEventRecord(start, 0);
//	hipEventSynchronize(start);  	// Needed?

	cpu_matrixadd(a,b,d,N);		// do calculation on host

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

	printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms);  // print out execution time
}

